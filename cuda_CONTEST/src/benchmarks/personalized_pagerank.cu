#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sstream>
#include "personalized_pagerank.cuh"

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

//////////////////////////////
//////////////////////////////

//Parallel reduction

__device__ void warp_reduce(volatile double* input, int thread_id) {
    int warp_size = 32;
    for (int i = warp_size; i > 0; i>>=1) {
        input[thread_id] += input[thread_id + i];
    }
}

__device__ void accumulate(double* input , int dim){
    int thread_id = threadIdx.x;
    if (dim > 32) {
        for (int i = dim/2; i > 32; i>>=1) {
            if (thread_id < i) {
                input[thread_id] += input[thread_id + i];
            }
            __syncthreads();
        }
    }

    if (thread_id < 32) {
        warp_reduce(input, thread_id);
    }

    __syncthreads();
}


// Write GPU kernel here!

__global__ void spmv_coo_gpu (const int* row_ids, const int* col_ids, const double* vals, const double* in_vec, double* out_vec , const int numVals) {
  for ( int i = threadIdx.x + blockIdx.x * blockDim.x ; i < numVals ; i += blockDim.x * gridDim.x ) {
    if ( i < numVals ) {
        atomicAdd(out_vec + row_ids[i], vals[i] * in_vec[col_ids[i]]);
    }
  }
}


/*** Dot Product Kernels  ***/
__global__ void dot_product_gpu (const int* vec1 , const double* vec2 , const int numVals , double* result){
    for(int i = threadIdx.x + blockIdx.x * blockDim.x ; i < numVals ; i += blockDim.x * gridDim.x ){
        if(i < numVals ){
            atomicAdd(result , vec1[i] * vec2[i]);
        }
    }

}

__global__ void dot_product_gpu_with_reduction (const int* vec1 , const double* vec2 , const int numVals , double* result) {
    extern __shared__ double tmp_res[];
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < numVals) {
        tmp_res[threadIdx.x] = vec1[id] * vec2[id];
    }
    __syncthreads();
    accumulate(tmp_res, blockDim.x);
    if (threadIdx.x == 0) {
        atomicAdd(result, tmp_res[0]);
    }
}


/*** AXPB Personalized Kernels***/

__global__ void  axpb_personalized_gpu(const double alpha , const double* prTmp, const double beta, const int personalizationVertex , double* result , const int numVals){
    double oneMinusalpha = 1 - alpha;
    for(int i = threadIdx.x + blockIdx.x * blockDim.x ; i < numVals ; i += blockDim.x * gridDim.x ){
        if(i < numVals){
            result[i] = alpha * prTmp[i] + beta + ((personalizationVertex == i) ? oneMinusalpha : 0.0);
        }
    }
}


/*** Euclidean Distance Kernels ***/

__global__ void euclidean_distance_gpu(const double* pr , const double* prTmp , double* err , const int numVals){
        for(int i = threadIdx.x + blockIdx.x * blockDim.x ; i < numVals ; i += blockDim.x * gridDim.x ){
            if(i < numVals){
                double tmp = pr[i] - prTmp[i];
                atomicAdd(err, tmp*tmp);
            }
        }
}

__global__ void euclidean_distance_gpu_with_reduction (const double* pr , const double* prTmp , double* err , const int numVals) {
    extern __shared__ double tmp_res[];
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < numVals) {
        double tmp = pr[id] - prTmp[id];
        tmp_res[threadIdx.x] = tmp*tmp;
    }
    __syncthreads();
    accumulate(tmp_res, blockDim.x);
    if (threadIdx.x == 0) {
        atomicAdd(err, tmp_res[0]);
    }
}

//////////////////////////////
//////////////////////////////

// CPU Utility functions;

int partition(std::vector<int>& col,std::vector<int>& row,std::vector<int>& vals, int start, int end){

	int pivot = col[start];

	int count = 0;
	for (int i = start + 1; i <= end; i++) {
		if (col[i] <= pivot)
			count++;
	}

	// Giving pivot element its correct position
	int pivotIndex = start + count;
	std::swap(col[pivotIndex], col[start]);
	std::swap(row[pivotIndex], row[start]);
	std::swap(vals[pivotIndex], vals[start]);

	// Sorting left and right parts of the pivot element
	int i = start, j = end;

	while (i < pivotIndex && j > pivotIndex) {

		while (col[i] <= pivot) {
			i++;
		}

		while (col[j] > pivot) {
			j--;
		}

		if (i < pivotIndex && j > pivotIndex) {
			std::swap(col[i], col[j]);
			std::swap(row[i], row[j]);
			std::swap(vals[i], vals[j]);
            i++;
            j--;
		}
	}

	return pivotIndex;
}

void quickSort(std::vector<int>& col ,std::vector<int>& row ,std::vector<int>& vals , int start, int end){

	// base case
	if (start >= end)
		return;

	// partitioning the array
	int p = partition(col,row,vals, start, end);

	// Sorting the left part
	quickSort(col,row,vals, start, p - 1);

	// Sorting the right part
	quickSort(col,row,vals, p + 1, end);
}


void PersonalizedPageRank::sort_scoo(){
    for(int i = 0; i < idx.size(); i++){
        int start = idx[i];
        int end = i == idx.size()-1 ? x.size() : idx[i+1];

        quickSort(x, y, val , start , end-1);

    }
}



//transform from coo to scoo,
//!requires a non sorted coo
void PersonalizedPageRank::coo_to_scoo(int slice_size){
    std::vector<int> col;
    std::vector<int> row;
    std::vector<double> vals;

    col.resize(E);
    row.resize(E);
    vals.resize(E);

    int ptr = 0;
    idx.resize(V / slice_size);
    idx[0] = 0; //first idx is always 0;

    for(int i = 0; i < V / slice_size; i++){
        for (int j = 0 ; j < E ; j++){
            if(y[j]>=(i * slice_size) && y[j] < (i*slice_size + slice_size)){
                col[ptr] = x[j];
                row[ptr] = y[j];
                vals[ptr] = val[j];
                ptr++;
            }
            idx[i+1] = ptr;
        }
    }

    x = col;
    y = row;
    val = vals;

}



// Read the input graph and initialize it;
void PersonalizedPageRank::initialize_graph() {
    
    bool sortMatrix = true;

    if(implementation == 2) sortMatrix = false;

    // Read the graph from an MTX file;
    int num_rows = 0;
    int num_columns = 0;
    read_mtx(graph_file_path.c_str(), &x, &y, &val,
        &num_rows, &num_columns, &E, // Store the number of vertices (row and columns must be the same value), and edges;
        true,                        // If true, read edges TRANSPOSED, i.e. edge (2, 3) is loaded as (3, 2). We set this true as it simplifies the PPR computation;
        false,                       // If true, read the third column of the matrix file. If false, set all values to 1 (this is what you want when reading a graph topology);
        debug,                 
        false,                       // MTX files use indices starting from 1. If for whatever reason your MTX files uses indices that start from 0, set zero_indexed_file=true;
        sortMatrix                         // If true, sort the edges in (x, y) order. If you have a sorted MTX file, turn this to false to make loading faster;
    );
    if (num_rows != num_columns) {
        if (debug) std::cout << "error, the matrix is not squared, rows=" << num_rows << ", columns=" << num_columns << std::endl;
        exit(-1);
    } else {
        V = num_rows;
    }
    if (debug) std::cout << "loaded graph, |V|=" << V << ", |E|=" << E << std::endl;

    // Compute the dangling vector. A vertex is not dangling if it has at least 1 outgoing edge;
    dangling.resize(V);
    std::fill(dangling.begin(), dangling.end(), 1);  // Initially assume all vertices to be dangling;
    for (int i = 0; i < E; i++) {
        // Ignore self-loops, a vertex is still dangling if it has only self-loops;
        if (x[i] != y[i]) dangling[y[i]] = 0;
    }
    // Initialize the CPU PageRank vector;
    pr.resize(V);
    pr_golden.resize(V);
    // Initialize the value vector of the graph (1 / outdegree of each vertex).
    // Count how many edges start in each vertex (here, the source vertex is y as the matrix is transposed);
    int *outdegree = (int *) calloc(V, sizeof(int));
    for (int i = 0; i < E; i++) {
        outdegree[y[i]]++;
    }
    // Divide each edge value by the outdegree of the source vertex;
    for (int i = 0; i < E; i++) {
        val[i] = 1.0 / outdegree[y[i]];  
    }
    free(outdegree);
}

//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;
void PersonalizedPageRank::alloc() {
    // Load the input graph and preprocess it;
    initialize_graph();

    //convert coo to scoo
    if( implementation == 2){
        coo_to_scoo();
    }
    
    // Allocate any GPU data here;
    CHECK(hipMalloc(&d_x , sizeof(int) * E););
    CHECK(hipMalloc(&d_y , sizeof(int) * E););
    CHECK(hipMalloc(&d_dangling , sizeof(int) * V););
    CHECK(hipMalloc(&d_val , sizeof(double) * E););
    CHECK(hipMalloc(&d_danglingFactor , sizeof(double)););
    CHECK(hipMalloc(&d_pr , sizeof(double) * V););
    CHECK(hipMalloc(&d_prTmp , sizeof(double) * V););
    CHECK(hipMalloc(&d_err , sizeof(double)););

    CHECK(hipMemcpy(d_x , x.data() , sizeof(int) * E , hipMemcpyHostToDevice););
    CHECK(hipMemcpy(d_y , y.data() , sizeof(int) * E , hipMemcpyHostToDevice););
    CHECK(hipMemcpy(d_val , val.data() , sizeof(double) * E , hipMemcpyHostToDevice););
    CHECK(hipMemcpy(d_dangling , dangling.data() , sizeof(int) * V , hipMemcpyHostToDevice););


}

// Initialize data;
void PersonalizedPageRank::init() {
    // Do any additional CPU or GPU setup here;
    blockNums = (V + block_size -1)/block_size;
    threadsPerBlockNums = block_size;
}

// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void PersonalizedPageRank::reset() {
   // Reset the PageRank vector (uniform initialization, 1 / V for each vertex);
   std::fill(pr.begin(), pr.end(), 1.0 / V); 
   // Generate a new personalization vertex for this iteration;
   personalization_vertex = rand() % V; 
   if (debug) std::cout << "personalization vertex=" << personalization_vertex << std::endl;

   // Do any GPU reset here, and also transfer data to the GPU;
    CHECK(hipMemcpy(d_pr , pr.data() , sizeof(double) * V , hipMemcpyHostToDevice););


}

void checkRes (double * vec1 ,double * vec2 , int size){
    for(unsigned int i = 0 ; i<size ; i++){
        if(std::abs(vec1[i] - vec2[i]) > std::pow(10, -4)) {
            std::cout << "ERROR " << i  << " " << vec1[i] << " " <<  vec2[i] << std::endl;
            exit(-1);
        }
    }

}


void PersonalizedPageRank::ppr_0 (int iter) {
    auto start_tmp = clock_type::now();

    // Do the GPU computation here, and also transfer results to the CPU;
    int numIter = 0;
    bool converged = false;

    dim3 blocks(blockNums , 1 , 1);
    dim3 threads(threadsPerBlockNums, 1 , 1);
    dim3 blocks_spmv((E + block_size -1)/block_size , 1 , 1);

    while(numIter < max_iterations && !converged ){
        double danglingFactor;
        double err;

        CHECK(hipMemset(d_prTmp , 0.0 , sizeof(double)*V););
        CHECK(hipMemset(d_err , 0.0 , sizeof(double)););
        CHECK(hipMemset(d_danglingFactor , 0.0 , sizeof(double)););
        
        spmv_coo_gpu<<<blocks_spmv , threads>>>(d_x, d_y, d_val ,d_pr , d_prTmp ,E);
        CHECK_KERNELCALL()


        dot_product_gpu<<<blocks , threads>>>(d_dangling , d_pr , V ,  d_danglingFactor);
        hipMemcpy(&danglingFactor , d_danglingFactor , sizeof(double) , hipMemcpyDeviceToHost);
        CHECK_KERNELCALL()
        
        axpb_personalized_gpu<<<blocks , threads>>>(alpha , d_prTmp , alpha * danglingFactor / V , personalization_vertex , d_prTmp , V);
        CHECK_KERNELCALL()

        euclidean_distance_gpu<<<blocks , threads>>>(d_pr, d_prTmp, d_err , V);
        hipMemcpy(&err , d_err , sizeof(double) , hipMemcpyDeviceToHost);
        CHECK_KERNELCALL()

        err = std::sqrt(err);
        converged = err <= convergence_threshold;

        CHECK(hipMemcpy(d_pr , d_prTmp , sizeof(double)*V , hipMemcpyDeviceToDevice));


        numIter++;

    }

    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        hipDeviceSynchronize();
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << double(exec_time) / 1000 << " ms, " << (sizeof(double) * N / (exec_time * 1e3)) << " GB/s" << std::endl;
    }

    CHECK(hipMemcpy(pr.data() , d_pr , sizeof(double) * V , hipMemcpyDeviceToHost));
}

void PersonalizedPageRank::ppr_1(int iter) {
    auto start_tmp = clock_type::now();

    // Do the GPU computation here, and also transfer results to the CPU;
    int numIter = 0;
    bool converged = false;

    dim3 blocks(blockNums , 1 , 1);
    dim3 threads(threadsPerBlockNums, 1 , 1);
    dim3 blocks_spmv((E + block_size -1)/block_size , 1 , 1);

    while(numIter < max_iterations && !converged ){
        double danglingFactor;
        double err;

        CHECK(hipMemset(d_prTmp , 0.0 , sizeof(double)*V););
        CHECK(hipMemset(d_err , 0.0 , sizeof(double)););
        CHECK(hipMemset(d_danglingFactor , 0.0 , sizeof(double)););
        
        spmv_coo_gpu<<<blocks_spmv , threads>>>(d_x, d_y, d_val ,d_pr , d_prTmp ,E);
        CHECK_KERNELCALL()


        dot_product_gpu_with_reduction<<<blocks , threads, block_size * sizeof(double)>>>(d_dangling , d_pr , V ,  d_danglingFactor);
        hipMemcpy(&danglingFactor , d_danglingFactor , sizeof(double) , hipMemcpyDeviceToHost);
        CHECK_KERNELCALL()
        
        axpb_personalized_gpu<<<blocks , threads>>>(alpha , d_prTmp , alpha * danglingFactor / V , personalization_vertex , d_prTmp , V);
        CHECK_KERNELCALL()

        euclidean_distance_gpu_with_reduction<<<blocks , threads, block_size * sizeof(double)>>>(d_pr, d_prTmp, d_err , V);
        hipMemcpy(&err , d_err , sizeof(double) , hipMemcpyDeviceToHost);
        CHECK_KERNELCALL()

        err = std::sqrt(err);
        converged = err <= convergence_threshold;

        CHECK(hipMemcpy(d_pr , d_prTmp , sizeof(double)*V , hipMemcpyDeviceToDevice));


        numIter++;

    }

    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        hipDeviceSynchronize();
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << double(exec_time) / 1000 << " ms, " << (sizeof(double) * N / (exec_time * 1e3)) << " GB/s" << std::endl;
    }

    CHECK(hipMemcpy(pr.data() , d_pr , sizeof(double) * V , hipMemcpyDeviceToHost));

}

void PersonalizedPageRank::execute(int iter) {
    
    switch (implementation)
    {
    case 0:
        ppr_0(iter);
        break;
    case 1:
        ppr_1(iter);
        break;    
    default:
        break;
    }

}

void PersonalizedPageRank::cpu_validation(int iter) {

    // Reset the CPU PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr_golden.begin(), pr_golden.end(), 1.0 / V);

    // Do Personalized PageRank on CPU;
    auto start_tmp = clock_type::now();
    personalized_pagerank_cpu(x.data(), y.data(), val.data(), V, E, pr_golden.data(), dangling.data(), personalization_vertex, alpha, 1e-6, 100);
    auto end_tmp = clock_type::now();
    auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
    std::cout << "exec time CPU=" << double(exec_time) / 1000 << " ms" << std::endl;

    // Obtain the vertices with highest PPR value;
    std::vector<std::pair<int, double>> sorted_pr_tuples = sort_pr(pr.data(), V);
    std::vector<std::pair<int, double>> sorted_pr_golden_tuples = sort_pr(pr_golden.data(), V);

    // Check how many of the correct top-20 PPR vertices are retrieved by the GPU;
    std::unordered_set<int> top_pr_indices;
    std::unordered_set<int> top_pr_golden_indices;
    int old_precision = std::cout.precision();
    std::cout.precision(4);
    int topk = std::min(V, topk_vertices);
    for (int i = 0; i < topk; i++) {
        int pr_id_gpu = sorted_pr_tuples[i].first;
        int pr_id_cpu = sorted_pr_golden_tuples[i].first;
        top_pr_indices.insert(pr_id_gpu);
        top_pr_golden_indices.insert(pr_id_cpu);
        if (debug) {
            double pr_val_gpu = sorted_pr_tuples[i].second;
            double pr_val_cpu = sorted_pr_golden_tuples[i].second;
            if (pr_id_gpu != pr_id_cpu) {
                std::cout << "* error in rank! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            } else if (std::abs(sorted_pr_tuples[i].second - sorted_pr_golden_tuples[i].second) > 1e-6) {
                std::cout << "* error in value! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            }
        }
    }
    std::cout.precision(old_precision);
    // Set intersection to find correctly retrieved vertices;
    std::vector<int> correctly_retrieved_vertices;
    set_intersection(top_pr_indices.begin(), top_pr_indices.end(), top_pr_golden_indices.begin(), top_pr_golden_indices.end(), std::back_inserter(correctly_retrieved_vertices));
    precision = double(correctly_retrieved_vertices.size()) / topk;
    if (debug) std::cout << "correctly retrived top-" << topk << " vertices=" << correctly_retrieved_vertices.size() << " (" << 100 * precision << "%)" << std::endl;
}

std::string PersonalizedPageRank::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(precision);
    } else {
        // Print the first few PageRank values (not sorted);
        std::ostringstream out;
        out.precision(3);
        out << "[";
        for (int i = 0; i < std::min(20, V); i++) {
            out << pr[i] << ", ";
        }
        out << "...]";
        return out.str();
    }
}

void PersonalizedPageRank::clean() {
    // Delete any GPU data or additional CPU data;
    hipFree(&d_x);
    hipFree(&d_y);
    hipFree(&d_val);
    hipFree(&d_dangling);
    hipFree(&d_danglingFactor);
    hipFree(&d_pr);
    hipFree(&d_prTmp);
    hipFree(&d_err);
}
